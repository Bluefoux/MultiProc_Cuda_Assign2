﻿
#include "hip/hip_runtime.h"



#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>

#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <chrono>


__global__ void oddeven(int* numbers)
{
    int temp;
    if (numbers[threadIdx.x] > numbers[threadIdx.x + 1]) {
        temp = numbers[threadIdx.x];
        numbers[threadIdx.x] = numbers[threadIdx.x + 1];
        numbers[threadIdx.x + 1] = temp;
    }
    __syncthreads();
}

void print_sort_status(int* numbers, int size)
{
    std::cout << "The input is sorted?: " << (std::is_sorted(&numbers[0], &numbers[size - 1]) == 0 ? "False" : "True") << std::endl;
}

int main()
{
    int size = 10; // Number of elements in the input

    // Initialize a vector with integers of value 0
    int* numbers;
    int* tmp;
    int* fin;
    int test;
    int k;
    srand(time(0));
    // Populate our vector with (pseudo)random numbers
    tmp = (int*)malloc(size * sizeof(int*));
    fin = (int*)malloc(size * sizeof(int*));

    for (k = 0; k < size; k++) {
        test = rand() % 100;
        tmp[k] = test;
        printf("rand = %d \n", tmp[k]);
    }


    hipMalloc((void**)&numbers, size * sizeof(int*));
    hipMemcpy(numbers, tmp, size * sizeof(int*), hipMemcpyHostToDevice);

    print_sort_status(tmp, size);
    auto start = std::chrono::steady_clock::now();
    for (int i = 0; i < size; i++) {
        for (int j = i % 2; j < size-1; j = j + 2) { // j = i%2
            oddeven <<< 1, i >>>(numbers);
        }
        hipDeviceSynchronize();
    }
    
    auto end = std::chrono::steady_clock::now();
    hipMemcpy(fin, numbers, size*sizeof(int*), hipMemcpyDeviceToHost);
    print_sort_status(fin, size);
    for (int l = 0; l < size; l++) {
        printf("fin = %d \n", fin[l]);
    }
    std::cout << "Elapsed time =  " << std::chrono::duration<double>(end - start).count() << " sec\n";
    //cudaMemcpy(fin,numbers,size*sizeof(int),cudaMemcpyHostToDevice);
    hipFree(numbers);
    return 0;
}