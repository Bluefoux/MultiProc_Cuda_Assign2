﻿
#include "hip/hip_runtime.h"



#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>

#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <chrono>


__global__ void oddeven(int* numbers, int size)
{
    int temp;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //numbers[i] = i;
    for (int j = i % 2; j < size; j = j + 2) {
        for (int k = 0; k < size/blockDim.x; k++) {
            i = i + (k * blockDim.x);
            if ((numbers[i] > numbers[i + 1]) && ((i % 2) == 0) && (i < size-1)) {
                temp = numbers[i];
                numbers[i] = numbers[i + 1];
                numbers[i + 1] = temp;
                //numbers[i] = i;
            }
            __syncthreads();
            if ((numbers[i] > numbers[i + 1]) && ((i % 2) != 0) && (i < size - 1)) {
                temp = numbers[i];
                numbers[i] = numbers[i + 1];
                numbers[i + 1] = temp;
                //numbers[i] = i;
            }
            __syncthreads();
        }
    }
}

void print_sort_status(int* numbers, int size)
{
    std::cout << "The input is sorted?: " << (std::is_sorted(&numbers[0], &numbers[size - 1]) == 0 ? "False" : "True") << std::endl;
}

int main()
{
    // Initialize a vector with integers of value 0
    int* numbers;
    int* tmp;
    int* fin;
    int test;
    int k;
    int size = 100000; // Number of elements in the input
    int var = std::min(size-1, 1024);
    //int var = 5;

    srand(time(0));
    // Populate our vector with (pseudo)random numbers
    tmp = (int*)malloc(size * sizeof(int*));
    fin = (int*)malloc(size * sizeof(int*));

    for (k = 0; k < size; k++) {
        test = rand();
        tmp[k] = test;
        //printf("tmp = %d \n", tmp[k]);
    }

    hipMalloc((void**)&numbers, size * sizeof(int*));
    hipMemcpy(numbers, tmp, size * sizeof(int*), hipMemcpyHostToDevice);
    print_sort_status(tmp, size);
    auto start = std::chrono::steady_clock::now();

    int threadsPerBlock(var);
    int numBlocks(size / threadsPerBlock);
    oddeven <<< numBlocks, threadsPerBlock >>> (numbers, size);
    hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();
    hipMemcpy(fin, numbers, size * sizeof(int*), hipMemcpyDeviceToHost);
    print_sort_status(fin, size);
    /*for (k = 0; k < size; k++) {
        printf("fin = %d \n", fin[k]);
    }*/
    std::cout << "Elapsed time =  " << std::chrono::duration<double>(end - start).count() << " sec\n";
    hipFree(numbers);
    return 0;
}