#include "hip/hip_runtime.h"
﻿/***************************************************************************
 *
 * Sequential version of Gauss-Jordan row reduction
 *
 ***************************************************************************/



    /***************************************************************************
 *
 * Sequential version of Gauss-Jordan row reduction
 *
 ***************************************************************************/


#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_cooperative_groups.h>


#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <stdlib.h>


#define MAX_SIZE 4096
#define BLOCKSIZE_x 10
#define BLOCKSIZE_y 10


typedef double matrix[MAX_SIZE][MAX_SIZE];

int	N;		/* matrix size		*/
int	maxnum;		/* max number of element*/
char* Init;		/* matrix init type	*/
int	PRINT;		/* print switch		*/
matrix	A;		/* matrix A		*/
double	b[MAX_SIZE];	/* vector b             */
double	y[MAX_SIZE];	/* vector y             */

/* forward declarations */
void work(void);
void Init_Matrix(void);
void Print_Matrix(void);
void Init_Default(void);
int Read_Options(int, char**);
int iDivUp(int hostPtr, int b);
__global__ void gauswork1(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k);
__global__ void gauswork2(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k);
__global__ void gauswork3(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k);
__global__ void gauswork4(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k);
__global__ void gauswork5(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k);
__global__ void gauswork6(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k);

int
main(int argc, char** argv)
{
    printf("Gauss Jordan\n");
    int i, timestart, timeend, iter;

    Init_Default();		/* Init default values	*/
    Read_Options(argc, argv);	/* Read arguments	*/
    Init_Matrix();		/* Init the matrix	*/
    work();
   if (PRINT == 1)
      Print_Matrix();
}

int iDivUp(int hostPtr, int b) { return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b); }

__global__ void gauswork1(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k) {

   
    int    tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int    tidy = blockIdx.y * blockDim.y + threadIdx.y;
   
    if ((tidx < MAX_SIZE) && (tidy < MAX_SIZE))
    {
       
        int i, j, p;

        double* row_a[MAX_SIZE];
        double* row_b;
        double* row_y;

 
        if (tidy == k && tidx > tidy && tidy < size && tidx < size) {
            for (int i = k; i < k + 1; i++) {
                row_a[i] = (double*)((char*)numbers + i * pitcher);
            }

            row_a[k][tidx] = row_a[k][tidx] / row_a[k][tidy];
        }
        __syncthreads();
    }
}
__global__ void gauswork2(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k) {


    int    tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int    tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if ((tidx < MAX_SIZE) && (tidy < MAX_SIZE))
    {

        int i, j, p;

        double* row_a[MAX_SIZE];
        double* row_b;
        double* row_y;


         if (tidy == k && tidx == k)
        {
             for (int i = k; i < k + 1; i++) {
                 row_a[i] = (double*)((char*)numbers + i * pitcher);
             }
             numy[k] = numb[k] / row_a[k][k];
             row_a[k][k] = 1.0;
        }
        __syncthreads();
    }
}

__global__ void gauswork3(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k) {

    int    tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int    tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if ((tidx < MAX_SIZE) && (tidy < MAX_SIZE))
    {

        int i, j, p;

        double* row_a[MAX_SIZE];
        double* row_b;
        double* row_y;

        
        if (tidy > k && tidx > k && tidx < size && tidy < size)
        {
            for (int i = tidy; i < size; i++)
            {
                row_a[i] = (double*)((char*)numbers + i * pitcher);
            }
            row_a[k] = (double*)((char*)numbers + k * pitcher);
            row_a[tidy][tidx] = row_a[tidy][tidx] - row_a[tidy][k] * row_a[k][tidx];
        }
        __syncthreads();
    }
}
__global__ void gauswork4(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k) {

    int    tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int    tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if ((tidx < MAX_SIZE) && (tidy < MAX_SIZE))
    {

        int i, j, p;

        double* row_a[MAX_SIZE];
        double* row_b;
        double* row_y;


        if (tidy > k && tidx == tidy && tidy < size)
        {
            for (int i = tidy; i < tidy+1; i++)
            {
                row_a[i] = (double*)((char*)numbers + i * pitcher);
            }

            numb[tidy] = numb[tidy] - row_a[tidy][k] * numy[k];
            row_a[tidy][k] = 0.0;

        }
        __syncthreads();
    }
}


__global__ void gauswork5(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k) {


    int    tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int    tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if ((tidx < MAX_SIZE) && (tidy < MAX_SIZE))
    {

        int i, j, p;

        double* row_a[MAX_SIZE];
        double* row_b;
        double* row_y;

        if (tidy < k && tidx > k && tidx <=size)
        {
            for (int i = 0; i < k + 1; i++) {
                row_a[i] = (double*)((char*)numbers + i * pitcher);
            }
            row_a[tidy][tidx] = row_a[tidy][tidx] - row_a[tidy][k] * row_a[k][tidx];
        }

    }
}
__global__ void gauswork6(double* numbers, size_t pitcher, double* numy, double* numb, int size, int k) {


    int    tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int    tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if ((tidx < MAX_SIZE) && (tidy < MAX_SIZE))
    {

        int i, j, p;

        double* row_a[MAX_SIZE];
        double* row_b;
        double* row_y;

        if (tidx == tidy && tidy < k)
        {
            for (int i = 0; i < k + 1; i++) {
                row_a[i] = (double*)((char*)numbers + i * pitcher);
            }
            numy[tidy] = numy[tidy] - row_a[tidy][k] * numy[k];
            row_a[tidy][k] = 0.0;
        }
        __syncthreads();
    }
}

void
work(void)
{
    int i;
    int j;
    int k;
    int size = N;
    int var = std::min(size - 1, 1024);
    double* numbers;
    double* numb;
    double* numy;
    
    size_t pitch;

    int kallex = iDivUp(size, BLOCKSIZE_x);
    int kalley = iDivUp(size, BLOCKSIZE_y);
    printf("kallex = %d \n", kallex);
    printf("kalley = %d \n", kalley);

    hipMallocPitch(&numbers, &pitch, MAX_SIZE * sizeof(double), MAX_SIZE);
    hipMalloc((void**)&numb, MAX_SIZE * sizeof(double*));
    hipMalloc((void**)&numy, MAX_SIZE * sizeof(double*));
    //hipMalloc((void**)&numbers, N * sizeof(double*));
    hipMemcpy2D(numbers, pitch, A, MAX_SIZE * sizeof(double), MAX_SIZE * sizeof(double), MAX_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(numb, b, MAX_SIZE * sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(numy, y, MAX_SIZE * sizeof(double*), hipMemcpyHostToDevice);
    
    //int threadsPerBlock(var);
    //int numBlocks(size);

    dim3 gridSize(iDivUp(size, BLOCKSIZE_x), iDivUp(size, BLOCKSIZE_y));
    dim3 blockSize(BLOCKSIZE_y, BLOCKSIZE_x);
    //dim3 gridSize(2, 2);
    //dim3 blockSize(10, 10);
    //printf("kallex = %d \n", kallex);
   // printf("kalley = %d \n", kalley);

    //int threadsPerBlock(var);
    //int numBlocks(size / threadsPerBlock);

    for (k = 0; k < N; k++) {
        int kalle = k;
        gauswork1 << <gridSize, blockSize >> > (numbers, pitch, numy, numb, N, kalle);
        hipDeviceSynchronize();

        gauswork2 << <gridSize, blockSize >> > (numbers, pitch, numy, numb, N, k);
        hipDeviceSynchronize();

        gauswork3 << <gridSize, blockSize >> > (numbers, pitch, numy, numb, N, k);
        hipDeviceSynchronize();
        gauswork4 << <gridSize, blockSize >> > (numbers, pitch, numy, numb, N, k);
        hipDeviceSynchronize();
        gauswork5 << <gridSize, blockSize >> > (numbers, pitch, numy, numb, N, k);
        hipDeviceSynchronize();
        gauswork6 << <gridSize, blockSize >> > (numbers, pitch, numy, numb, N, k);
        hipDeviceSynchronize();
    }

    //gauswork << <numBlocks, threadsPerBlock >> > (numbers, pitch, numy, numb);
    hipDeviceSynchronize();
    //gauswork << <numBlocks, threadsPerBlock >> > (numbers, pitch);
    //hipDeviceSynchronize();
    hipMemcpy2D(A, MAX_SIZE * sizeof(double), numbers, pitch, MAX_SIZE * sizeof(double), MAX_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(b, numb, size * sizeof(double*), hipMemcpyDeviceToHost);
    hipMemcpy(y, numy, size * sizeof(double*), hipMemcpyDeviceToHost);
    hipFree(numbers);
}

void
Init_Matrix()
{
    int i, j;
    N = 1000;
    printf("\nsize      = %dx%d ", N, N);
    printf("\nmaxnum    = %d \n", maxnum);
    printf("Init	  = %s \n", Init);
    printf("Initializing matrix...");

    if (strcmp(Init, "rand") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = (double)(rand() % maxnum) + 5.0;
                else
                    A[i][j] = (double)(rand() % maxnum) + 1.0;
            }
        }
    }
    if (strcmp(Init, "fast") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = 5.0;
                else
                    A[i][j] = 2.0;
            }
        }
    }

    /* Initialize vectors b and y */
    for (i = 0; i < N; i++) {
        b[i] = 2.0;
        y[i] = 1.0;
    }

    printf("done \n\n");
    //if (PRINT == 1)
      //  Print_Matrix();
}

void
Print_Matrix()
{
    int i, j;

    printf("Matrix A:\n");
    for (i = 0; i < N; i++) {
        printf("[");
        for (j = 0; j < N; j++)
            printf(" %5.2f,", A[i][j]);
        printf("]\n");
    }
    printf("Vector y:\n[");
    for (j = 0; j < N; j++) {
        printf(" %5.2f,", y[j]);
    }
    printf("]\n");
    printf("Vector b:\n[");
    for (j = 0; j < N; j++) {
        printf(" %5.2f,", b[j]);
    }
    printf("]\n");
    printf("\n\n");
}

void
Init_Default()
{
    N = 2048;
    Init = "fast";
    maxnum = 15.0;
    PRINT = 1;
}

int
Read_Options(int argc, char** argv)
{
    char* prog;

    prog = *argv;
    while (++argv, --argc > 0)
        if (**argv == '-')
            switch (*++ * argv) {
            case 'n':
                --argc;
                N = atoi(*++argv);
                break;
            case 'h':
                printf("\nHELP: try sor -u \n\n");
                exit(0);
                break;
            case 'u':
                printf("\nUsage: gaussian [-n problemsize]\n");
                printf("           [-D] show default values \n");
                printf("           [-h] help \n");
                printf("           [-I init_type] fast/rand \n");
                printf("           [-m maxnum] max random no \n");
                printf("           [-P print_switch] 0/1 \n");
                exit(0);
                break;
            case 'D':
                printf("\nDefault:  n         = %d ", N);
                printf("\n          Init      = rand");
                printf("\n          maxnum    = 5 ");
                printf("\n          P         = 0 \n\n");
                exit(0);
                break;
            case 'I':
                --argc;
                Init = *++argv;
                break;
            case 'm':
                --argc;
                maxnum = atoi(*++argv);
                break;
            case 'P':
                --argc;
                PRINT = atoi(*++argv);
                break;
            default:
                printf("%s: ignored option: -%s\n", prog, *argv);
                printf("HELP: try %s -u \n\n", prog);
                break;
            }
}